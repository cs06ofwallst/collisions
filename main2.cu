#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "collisions_continuous.cu"
__device__ float Side(float3 a, float3 b, float3 c, float3 l) {
    // (a - c) and (b - c)
    float3 ac = make_float3(a.x - c.x, a.y - c.y, a.z - c.z);
    float3 bc = make_float3(b.x - c.x, b.y - c.y, b.z - c.z);

    // cross product (a - c) x (b - c)
    float3 cross_product = make_float3(
        ac.y * bc.z - ac.z * bc.y,
        ac.z * bc.x - ac.x * bc.z,
        ac.x * bc.y - ac.y * bc.x
    );

    float side = cross_product.x * l.x + cross_product.y * l.y + cross_product.z * l.z;
    return side;
}

__device__ int SideSign(float3 a, float3 b, float3 c, float3 l) {
    float side = Side(a, b, c, l);
    if (side > 0) {
        return 1;
    } else if (side < 0) {
        return -1;
    } else {
        return 0;
    }
}
__global__ bool UnprojectedContourTest(float3* vertices, float3 coneAxis) {
    int numVertices = sizeof(vertices)/sizeof(float3);
    int i = threadIdx.x;
    if (i >= numVertices - 1) return;

    // init intersection nr to 0
    __shared__ int intNum;
    if (threadIdx.x == 0) intNum = 0;
    __syncthreads();

    // determine self intersection
    __shared__ int res;
    if (threadIdx.x == 0) res = 1;  // assume true
    __syncthreads();

    // params
    float3 o = make_float3(0, 0, 0);
    for (int j = 0; j < numVertices; j++) {
        o.x += vertices[j].x;
        o.y += vertices[j].y;
        o.z += vertices[j].z;
    }
    o.x /= numVertices;
    o.y /= numVertices;
    o.z /= numVertices;

    // r is the axis perpendicular to the cone axis
    float3 r = make_float3(1, 0, 0);
    //check parallel to (0,1,0)
    if (coneAxis.x == 0.0f && coneAxis.z == 0.0f && coneAxis.y != 0.0f) {
        float3 r = make_float3(1, 0, 0);
    } else {
        float3 r1 = make_float3(0, 1, 0);
        float3 r = make_float3(0, coneAxis.y, 0);
    }
    // side sign at 1st contour segment
    int s0 = SideSign(o, vertices[0], vertices[1], coneAxis);  // TODO: implement
    if (s0 == 0) {
        return false;
    }
    __syncthreads();
   int n = 0;
   for (int i=0; i<numVertices; i++) {
    
    // side + intersection test on each contour segment in parallel
        // side
        int s1 = SideSign(o, vertices[i], vertices[(i + 1) % numVertices], coneAxis); 
        if (s0 != s1) {
            return false;  
        }

        // intersection
        int s2 = SideSign(vertices[i], o, o + r, coneAxis);  
        int s3 = SideSign(vertices[i+1], o, o + r, coneAxis);
        if (s3 == 0 || s2 == 0) {
            return false;  
        }

        if (s3 == s0 && s3 != s2) {
            n++;  // atomic operation for shared variable
            if (n > 1) {
                return false;
            }
        }
    
     __syncthreads();
   }
  return true;
}

// example main
int main() {
    int numVertices = 4;

    // cone axis and vertices, does it work like this?
    float3 coneAxis = make_float3(0, 0, 1);
    float3 vertices[] = {
        make_float3(1, 0, 0),
        make_float3(0, 1, 0),
        make_float3(-1, 0, 0),
        make_float3(0, -1, 0)
    };

    // device
    float3* d_vertices;
    int* d_result;
    int result = 0;

    // alloc mem
    hipMalloc((void**)&d_vertices, numVertices * sizeof(float3));
    hipMalloc((void**)&d_result, sizeof(int));

    // copy to device
    hipMemcpy(d_vertices, vertices, numVertices * sizeof(float3), hipMemcpyHostToDevice);

    // kernel
    UnprojectedContourTest<<<1, numVertices>>>(d_vertices, numVertices, coneAxis, d_result);

    // back to host
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_vertices);
    hipFree(d_result);

    // output
    if (result == 1) {
        printf("No self-intersection on the projected contour.\n");
    } else {
        printf("Self-intersection detected on the projected contour.\n");
    }

    return 0;
}

// BVH Node structure
struct BVHNode {
    bool isLeaf;         // leaf?
    float apexAngle;     // apex andgle of normal cone
    float3* vertices;    // vertices on node
    int numVertices;     // nr of vertices
    BVHNode* leftChild;  // reference
    BVHNode* rightChild; // reference
    float3 coneAxis;     // normal cone axis
};

// idk
__device__ bool UnprojectedContourTest(float3* vertices, int numVertices, float3 coneAxis, float apexAngle);
__device__ void Collide(BVHNode* leftNode, BVHNode* rightNode);

// recursive function
__device__ void SelfCollide(BVHNode* N) {
    // base (leaf -> terminate)
    if (N->isLeaf) {
        return;  // Traversal terminated
    }

    // is the apex angle of the normal cone less than pi
    if (N->apexAngle < M_PI) {
        // unprojected contour test
        if (UnprojectedContourTest(N->vertices, N->numVertices, N->coneAxis, N->apexAngle)) {
            return;  // no self-collisions
        }
    }

    // check left and right children
    SelfCollide(N->leftChild);
    SelfCollide(N->rightChild);

    // collisions between left and right children
    Collide(N->leftChild, N->rightChild);
}

// handle collisions between two BVH nodes
__device__ void Collide(BVHNode* leftNode, BVHNode* rightNode) {
    // TODO
}

// init SelfCollide on a BVH root
__global__ void SelfCollideKernel(BVHNode* root) {
    // Launch the SelfCollide recursive function on the BVH root
    SelfCollide(root);
}

int main() {
    // example
    BVHNode root;
    BVHNode leftChild, rightChild;

    // build
    root.isLeaf = false;
    root.apexAngle = M_PI / 4;  
    root.coneAxis = make_float3(0, 0, 1);  
    root.leftChild = &leftChild;
    root.rightChild = &rightChild;

    leftChild.isLeaf = true;  // left is leaf
    rightChild.isLeaf = true;  // right is leaf

    //mem alloc
    BVHNode* d_root;
    hipMalloc((void**)&d_root, sizeof(BVHNode));

    // copy to device
    hipMemcpy(d_root, &root, sizeof(BVHNode), hipMemcpyHostToDevice);

    // 1 thread (recursive)
    SelfCollideKernel<<<1, 1>>>(d_root);

    // synchr
    hipDeviceSynchronize();

    // free
    hipFree(d_root);

    return 0;
}

__device__ int CSideSign1(float3 o, float3 v1, float3 v2, float3 l) {
    // TODO
    return 0; 
}
__device__ int CSideSign2(float3 o, float3 v1, float3 v2, float3 l) {
    // TODO
    return 0; 
}
__global__ void UnprojectedContourTestForCCD(float3* vertices, float3 coneAxis) {
    int numVertices = sizeof(vertices)/sizeof(float3);
    int i = threadIdx.x;
    if (i >= numVertices - 1) return;

    // init intersection nr to 0
    __shared__ int intNum;
    if (threadIdx.x == 0) intNum = 0;
    __syncthreads();

    // determine self intersection
    __shared__ int res;
    if (threadIdx.x == 0) res = 1;  // assume true
    __syncthreads();

    // params
    float3 o = make_float3(0, 0, 0);
    for (int j = 0; j < numVertices; j++) {
        o.x += vertices[j].x;
        o.y += vertices[j].y;
        o.z += vertices[j].z;
    }
    o.x /= numVertices;
    o.y /= numVertices;
    o.z /= numVertices;

    // r is the axis perpendicular to the cone axis
    float3 r = make_float3(1, 0, 0);
    //check parallel to (0,1,0)
    if (coneAxis.x == 0.0f && coneAxis.z == 0.0f && coneAxis.y != 0.0f) {
        float3 r = make_float3(1, 0, 0);
    } else {
        float3 r1 = make_float3(0, 1, 0);
        float3 r = make_float3(0, coneAxis.y, 0);
    }
    // side sign at 1st contour segment
    int s0 = CSideSign1(o, vertices[0], vertices[1], coneAxis);  // TODO: implement
    if (s0 == 0) {
        return false;
    }
    __syncthreads();
   int n = 0;
   for (int i=0; i<numVertices; i++) {
    
    // side + intersection test on each contour segment in parallel
        // side
        int s1 = CSideSign1(o, vertices[i], vertices[(i + 1) % numVertices], coneAxis); 
        if (s0 != s1) {
            return false;  
        }

        // intersection
        int s2 = CSideSign2(vertices[i], o, o + r, coneAxis);  
        int s3 = CSideSign2(vertices[i+1], o, o + r, coneAxis);
        if (s3 == 0 || s2 == 0) {
            return false;  
        }

        if (s3 == s0 && s3 != s2) {
            n++;  // atomic operation for shared variable
            if (n > 1) {
                return false;
            }
        }
    
     __syncthreads();
   }
  return true;
}

int main() {
    // jsut another example
    int numVertices = 5;
    float alpha = 45.0f;  // apex angle
    float3 l = make_float3(0, 1, 0);  // axis of the cone

    float3 h_vertices[] = {
        make_float3(1, 0, 0),
        make_float3(0, 1, 0),
        make_float3(-1, 0, 0),
        make_float3(0, -1, 0),
        make_float3(0.5f, 0.5f, 0)
    };

    float3* d_vertices;
    bool* d_result;
    bool h_result;

    hipMalloc(&d_vertices, numVertices * sizeof(float3));
    hipMalloc(&d_result, sizeof(bool));

    // to device
    hipMemcpy(d_vertices, h_vertices, numVertices * sizeof(float3), hipMemcpyHostToDevice);

    // kernel
    UnprojectedContourTestForCCD<<<1, 1>>>(d_vertices, numVertices, alpha, l, d_result);

    // back
    hipMemcpy(&h_result, d_result, sizeof(bool), hipMemcpyDeviceToHost);

    // check
    if (h_result) {
        printf("No self-intersection on the projected contour.\n");
    } else {
        printf("Self-intersection detected or result is undetermined.\n");
    }

    hipFree(d_vertices);
    hipFree(d_result);

    return 0;
}

struct NormalCone {
    float apexAngle;
    // ...
};

struct BVTTFront {
};

__device__ bool IsLeaf(const BVHNode* node) {
    // TODO: deltermine if it's a leaf
    return node->isLeaf;
}

__device__ void FrontTracking(const BVTTFront* frontN) {
    // TODO: impl front tracking
}

__device__ void SelfCollideWithGuidedFrontTracking(BVHNode* N, BVTTFront* FrontN, NormalCone* CN) {
    // leaf?
    if (IsLeaf(N)) {
        return;  // done
    }

    // apex angle
    if (CN->apexAngle < M_PI) {
        // unprojected contour test
        if (UnprojectedContourTest(CN)) {
            return;  // free
        }
    }

    // recurse
    SelfCollideWithGuidedFrontTracking(N->leftChild, FrontN, CN);
    SelfCollideWithGuidedFrontTracking(N->rightChild, FrontN, CN);

    Collide(N->leftChild, N->rightChild);
}
